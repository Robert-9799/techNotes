#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define n 6

int main(int argc, char const *argv[]) {
  hipError_t cudaStat;
  hipblasStatus_t Stat;
  hipblasHandle_t handle;
  int j;
  float *x;
  x = (float*)malloc(n*sizeof(float));
  for (j = 0; j < n; j++)
    x[j] = (float)j;
  printf("x: ");
  for (j = 0; j < n; j++)
    printf("%4.0f, ", x[j]);
  printf("\n");

  float *d_x;
  cudaStat = hipMalloc(&d_x, n*sizeof(float));
  Stat = hipblasCreate(&handle);
  Stat = hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
  int result;

  Stat = hipblasIsamax(handle, n, d_x, 1, &result);
  printf("max |x[i]|:%4.0f\n", fabs(x[result-1]));
  hipFree(d_x);
  hipblasDestroy(handle);
  free(x);
  return EXIT_SUCCESS;
}
