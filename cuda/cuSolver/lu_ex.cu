#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsolver.h>

#include <gsl/gsl_linalg.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_vector.h>

void printMatrix(int m, int n, const double *A, int lda, const char* name)
{
  for (int row = 0; row < m; row++)
  {
    for (int col = 0; col < n; col++)
    {
      double Areg = A[row+col*lda];
      printf("%s(%d, %d) = %f\n", name, row+1, col+1, Areg);
    }
  }
}

// solve Ax = B
//
// where dim(A) = m*m
//       dim(B) = m*1
int cuda_LU_solve(const double *A, const int m, double *B)
{
  hipsolverHandle_t cusolverH = NULL;
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
  hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
  hipError_t cudaStat1 = hipSuccess;
  hipError_t cudaStat2 = hipSuccess;
  hipError_t cudaStat3 = hipSuccess;
  hipError_t cudaStat4 = hipSuccess;

  const int lda = m;
  const int ldb = m;
  const int nrhs = 1;
  //
  //     | 1 2 3 |
  // A = | 4 5 6 |
  //     | 2 1 1 |
  //
  // x = (1, 1, 1)'
  // b = (6, 15, 4)'
  //

  //double A[lda*m] = {1, 4, 2, 2, 5, 1, 3, 6, 1};
  //double B[ldb*nrhs] = {6, 15, 4};

  double *d_A = NULL; //linear memory of GPU
  //double *d_tau = NULL;
  int *devIpiv = NULL;
  double *d_B = NULL;
  int *devInfo = NULL;
  double *d_work = NULL;
  int lwork = 0;
  int info_gpu = 0;

  printf("A = (matlab base-1)\n");
  printMatrix(m, m, A, lda, "A");
  printf("=====\n");
  printf("B = (matlab base-1)\n");
  printMatrix(m, nrhs, B, ldb, "B");
  printf("=====\n");

  // step 1: create cusolver/cublas handle
  cusolver_status = hipsolverDnCreate(&cusolverH);
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

  cublas_status = hipblasCreate(&cublasH);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

  // step 2: copy A and B to device
  cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double) * lda * m);
  cudaStat2 = hipMalloc ((void**)&devIpiv, sizeof(int) * m);
  cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(double) * ldb * nrhs);
  cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);
  assert(hipSuccess == cudaStat4);

  cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m   , hipMemcpyHostToDevice);
  cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);

  // step 3: query working space of getrf and getrs
  cusolver_status = hipsolverDnDgetrf_bufferSize(cusolverH,
                      m,
                      m,
                      d_A,
                      lda,
                      &lwork );

  assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
  cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
  assert(hipSuccess == cudaStat1);

  // step 4: compute LU decomposition

  cusolver_status = hipsolverDnDgetrf(cusolverH,
           m,
           m,
           d_A,
           lda,
           d_work,
           devIpiv,
           devInfo );

  cudaStat1 = hipDeviceSynchronize();
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
  assert(hipSuccess == cudaStat1);

  // check if LU is good or not
  cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);
  printf("after getrf: info_gpu = %d\n", info_gpu);
  if (info_gpu < 0)
    printf("ERROR: the %d-th parameter is wrong.\n", abs(info_gpu));
  else if (info_gpu > 0)
    printf("ERROR: U(%d, %d) = 0\n", info_gpu, info_gpu);

  assert(0 == info_gpu);

  // setp 5: solve Ax = B
  cusolver_status = hipsolverDnDgetrs(cusolverH,
           HIPBLAS_OP_N,
           m,
           nrhs,
           d_A,
           lda,
           devIpiv,
           d_B,
           ldb,
           devInfo );

  cudaStat1 = hipDeviceSynchronize();
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
  assert(hipSuccess == cudaStat1);

  // check if LU is good or not
  cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);

  printf("after getrs: info_gpu = %d\n", info_gpu);
  if (info_gpu < 0)
    printf("ERROR: the %d-th parameter is wrong.\n", abs(info_gpu));

  assert(0 == info_gpu);


  cudaStat1 = hipMemcpy(B, d_B, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);

  printf("X = (matlab base-1)\n");
  printMatrix(m, nrhs, B, ldb, "X");

  // free resources
  if (d_A) hipFree(d_A);
  if (devIpiv) hipFree(devIpiv);
  if (d_B) hipFree(d_B);
  if (devInfo) hipFree(devInfo);
  if (d_work) hipFree(d_work);

  return 0;
}

int main(int argc, char const *argv[]) {
  double a_data[] = { 0.18,
0.41,
0.14,
0.51,
0.60,
0.24,
0.30,
0.13,
0.57,
0.99,
0.97,
0.19,
0.96,
0.58,
0.66,
0.85 };
double b_data[] = { 1.0, 2.0, 3.0, 4.0 };

  gsl_matrix_view m = gsl_matrix_view_array(a_data, 4, 4);
  gsl_vector_view b = gsl_vector_view_array(b_data, 4);
  printMatrix(4, 4, (&m.matrix)->data, 4, "A");
//  double A[] = {1, 4, 2, 2, 5, 1, 3, 6, 1};
//  double B[] = {6, 15, 4};
  cuda_LU_solve((&m.matrix)->data, 4, (&b.vector)->data);
  printMatrix(4, 1, (&b.vector)->data, 4, "x");
  return 0;
}
